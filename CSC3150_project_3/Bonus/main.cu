#include "hip/hip_runtime.h"
﻿#include "virtual_memory.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <inttypes.h>
#include <stdio.h>
#include <stdlib.h>

#define DATAFILE "./data.bin"
#define OUTFILE "./snapshot.bin"

// page size is 32bytes
#define PAGE_SIZE (1 << 5)
// 16 KB in page table
#define INVERT_PAGE_TABLE_SIZE (1 << 14)
// 32 KB in shared memory
#define PHYSICAL_MEM_SIZE (1 << 15)
// 128 KB in global memory
#define STORAGE_SIZE (1 << 17)

//// count the pagefault times
__device__ __managed__ int pagefault_num = 0;
__device__ __managed__ u32 ptCounter[1024] = {0};

// the thread id of the kernel thread
__device__ __managed__ uchar threadId;

// data input and output
__device__ __managed__ uchar results[STORAGE_SIZE]; // 128 KB
__device__ __managed__ uchar input[STORAGE_SIZE];

// memory allocation for virtual_memory
// secondary memory
__device__ __managed__ uchar storage[STORAGE_SIZE];
// page table & page table counter
extern __shared__ u32 pt[];

__device__ void user_program(VirtualMemory *vm, uchar *input, uchar *results,
                             int input_size);

__global__ void mykernel(int input_size) {
    // memory allocation for virtual_memory
    // take shared memory as physical memory
    __shared__ uchar data[PHYSICAL_MEM_SIZE];

    // execute the threads: 0->1->2->3
    if (threadIdx.x == 0) { // thread #0
        printf("now is the threadx:%d\n", threadIdx.x);
        VirtualMemory vm;
        vm_init(&vm, data, storage, pt, &pagefault_num, ptCounter, PAGE_SIZE,
            INVERT_PAGE_TABLE_SIZE, PHYSICAL_MEM_SIZE, STORAGE_SIZE,
            PHYSICAL_MEM_SIZE / PAGE_SIZE, 0);

        // user program the access pattern for testing paging
        user_program(&vm, input, results, input_size);
    }
    __syncthreads();
    if (threadIdx.x == 1) { // thread #1
        printf("now is the threadx:%d\n", threadIdx.x);
        VirtualMemory vm;
        vm_init(&vm, data, storage, pt, &pagefault_num, ptCounter, PAGE_SIZE,
            INVERT_PAGE_TABLE_SIZE, PHYSICAL_MEM_SIZE, STORAGE_SIZE,
            PHYSICAL_MEM_SIZE / PAGE_SIZE, 1);

        // user program the access pattern for testing paging
        user_program(&vm, input, results, input_size);
    }
    __syncthreads();
    if (threadIdx.x == 2) { // thread #2
        printf("now is the threadx:%d\n", threadIdx.x);
        VirtualMemory vm;
        vm_init(&vm, data, storage, pt, &pagefault_num, ptCounter, PAGE_SIZE,
            INVERT_PAGE_TABLE_SIZE, PHYSICAL_MEM_SIZE, STORAGE_SIZE,
            PHYSICAL_MEM_SIZE / PAGE_SIZE, 2);

        // user program the access pattern for testing paging
        user_program(&vm, input, results, input_size);
    }
    __syncthreads();
    if (threadIdx.x == 3) { // thread #3
        printf("now is the threadx:%d\n", threadIdx.x);
        VirtualMemory vm;
        vm_init(&vm, data, storage, pt, &pagefault_num, ptCounter, PAGE_SIZE,
            INVERT_PAGE_TABLE_SIZE, PHYSICAL_MEM_SIZE, STORAGE_SIZE,
            PHYSICAL_MEM_SIZE / PAGE_SIZE, 3);

        // user program the access pattern for testing paging
        user_program(&vm, input, results, input_size);
    }
}

__host__ void write_binaryFile(char *fileName, void *buffer, int bufferSize) {
  FILE *fp;
  fp = fopen(fileName, "wb");
  fwrite(buffer, 1, bufferSize, fp);
  fclose(fp);
}

__host__ int load_binaryFile(char *fileName, void *buffer, int bufferSize) {
  FILE *fp;

  fp = fopen(fileName, "rb");
  if (!fp) {
    printf("***Unable to open file %s***\n", fileName);
    exit(1);
  }

  // Get file length
  fseek(fp, 0, SEEK_END);
  int fileLen = ftell(fp);
  fseek(fp, 0, SEEK_SET);

  if (fileLen > bufferSize) {
    printf("****invalid testcase!!****\n");
    printf("****software warrning: the file: %s size****\n", fileName);
    printf("****is greater than buffer size****\n");
    exit(1);
  }

  // Read file contents into buffer
  fread(buffer, fileLen, 1, fp);
  fclose(fp);

  return fileLen;
}

int main() {
  hipError_t cudaStatus;
  int input_size = load_binaryFile(DATAFILE, input, STORAGE_SIZE);

  /* Launch kernel function in GPU, with single thread
  and dynamically allocate INVERT_PAGE_TABLE_SIZE bytes of share memory,
  which is used for variables declared as "extern __shared__" */
  mykernel<<<1, 4, INVERT_PAGE_TABLE_SIZE>>>(input_size);

  cudaStatus = hipGetLastError();
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "mykernel launch failed: %s\n",
            hipGetErrorString(cudaStatus));
    return 0;
  }

  printf("input size: %d\n", input_size);

  hipDeviceSynchronize();
  hipDeviceReset();

  write_binaryFile(OUTFILE, results, input_size);

  printf("pagefault number is %d\n", pagefault_num);

  return 0;
}
